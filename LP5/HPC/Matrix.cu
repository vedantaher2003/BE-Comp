
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;
using namespace std::chrono;

__global__ void multiply(int* A, int* B, int* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < K) {
        int sum = 0;
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * K + col];
        }
        C[row * K + col] = sum;
    }
}

void initialize(int* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        cout << "Enter element " << i + 1 << ": ";
        cin >> matrix[i];
    }
}

void print(int* matrix, int rows, int cols) {
    for (int row = 0; row < rows; row++) {
        for (int col = 0; col < cols; col++) {
            cout << matrix[row * cols + col] << " ";
        }
        cout << '\n';
    }
    cout << '\n';
}

void sequentialMultiply(int* A, int* B, int* C, int M, int N, int K) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            int sum = 0;
            for (int k = 0; k < N; k++) {
                sum += A[i * N + k] * B[k * K + j];
            }
            C[i * K + j] = sum;
        }
    }
}

int main() {
    int M, N, K;
    cout << "Enter the number of rows and columns of the first matrix: ";
    cin >> M >> N;
    cout << "Enter the number of columns of the second matrix: ";
    cin >> K;

    int* A, * B, * C;

    int matrixSize = M * K;
    size_t matrixBytes = matrixSize * sizeof(int);

    A = new int[M * N];
    B = new int[N * K];
    C = new int[M * K];

    initialize(A, M, N);
    initialize(B, N, K);

    cout << "Matrix A: \n";
    print(A, M, N);

    cout << "Matrix B: \n";
    print(B, N, K);

    int* X, * Y, * Z;
    hipMalloc(&X, M * N * sizeof(int));
    hipMalloc(&Y, N * K * sizeof(int));
    hipMalloc(&Z, M * K * sizeof(int));

    hipMemcpy(X, A, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Y, B, N * K * sizeof(int), hipMemcpyHostToDevice);

    int THREADS = 16;
    int BLOCKS = (M + THREADS - 1) / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    // Sequential multiplication
    auto start = high_resolution_clock::now();
    sequentialMultiply(A, B, C, M, N, K);
    auto stop = high_resolution_clock::now();
    auto seq_duration = duration_cast<microseconds>(stop - start);

    cout << "Sequential Multiplication of matrix A and B: \n";
    print(C, M, K);

    // Parallel multiplication
    start = high_resolution_clock::now();
    multiply<<<blocks, threads>>>(X, Y, Z, M, N, K);
    hipMemcpy(C, Z, M * K * sizeof(int), hipMemcpyDeviceToHost);
    stop = high_resolution_clock::now();
    auto par_duration = duration_cast<microseconds>(stop - start);

    cout << "Parallel Multiplication of matrix A and B: \n";
    print(C, M, K);

    cout << "Sequential Multiplication Time: " << seq_duration.count() << " microseconds" << endl;
    cout << "Parallel Multiplication Time: " << par_duration.count() << " microseconds" << endl;

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}
